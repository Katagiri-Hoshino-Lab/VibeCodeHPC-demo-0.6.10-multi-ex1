// GEMM CUDA v1.5.0 - Maximum register utilization and aggressive unrolling
// Targeting 60% efficiency with enhanced optimization techniques
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", \
                __FILE__, __LINE__, hipGetErrorString(error)); \
        exit(1); \
    } \
} while(0)

// v1.5.0: Larger tiles for better arithmetic intensity
#define BLOCK_M 128
#define BLOCK_N 128  
#define BLOCK_K 8
#define THREAD_M 8
#define THREAD_N 8

// Read-only cache optimization
__device__ __forceinline__ double ldg_double(const double* ptr) {
#if __CUDA_ARCH__ >= 350
    return __ldg(ptr);
#else
    return *ptr;
#endif
}

// FMA optimization
__device__ __forceinline__ double fma_double(double a, double b, double c) {
    return fma(a, b, c);
}

__global__ void __launch_bounds__(256, 2) gemm_kernel_optimized(
    int M, int N, int K,
    double alpha, const double* __restrict__ A, int lda,
    const double* __restrict__ B, int ldb,
    double beta, double* __restrict__ C, int ldc) {
    
    // Double buffering for overlapping computation and memory access
    __shared__ __align__(16) double As[2][BLOCK_M][BLOCK_K + 1];  // Padding to avoid bank conflicts
    __shared__ __align__(16) double Bs[2][BLOCK_K][BLOCK_N + 1];
    
    // Thread and block indices
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;
    
    // Warp configuration for optimal memory access
    const int warpId = tid / 32;
    const int laneId = tid % 32;
    const int warpRow = warpId / 2;
    const int warpCol = warpId % 2;
    
    // Thread tile position within warp
    const int threadRowInWarp = (laneId / 4) * THREAD_M;
    const int threadColInWarp = (laneId % 4) * THREAD_N * 2;
    
    // Global position
    const int globalRow = by * BLOCK_M + warpRow * 32 + threadRowInWarp;
    const int globalCol = bx * BLOCK_N + warpCol * 64 + threadColInWarp;
    
    // Register file for maximum accumulation
    double acc[THREAD_M][THREAD_N];
    
    // Initialize accumulators
    #pragma unroll
    for (int i = 0; i < THREAD_M; i++) {
        #pragma unroll
        for (int j = 0; j < THREAD_N; j++) {
            acc[i][j] = 0.0;
        }
    }
    
    // Pointers to global memory
    const double* A_global = A + by * BLOCK_M * lda;
    const double* B_global = B + bx * BLOCK_N;
    
    // Double buffering indices
    int write_buf = 0;
    int read_buf = 1;
    
    // Prefetch first tile
    // Load A tile
    #pragma unroll 4
    for (int i = tid; i < BLOCK_M * BLOCK_K; i += blockDim.x * blockDim.y) {
        int row = i / BLOCK_K;
        int col = i % BLOCK_K;
        if (by * BLOCK_M + row < M && col < K) {
            As[write_buf][row][col] = ldg_double(&A_global[row * lda + col]);
        } else {
            As[write_buf][row][col] = 0.0;
        }
    }
    
    // Load B tile
    #pragma unroll 4
    for (int i = tid; i < BLOCK_K * BLOCK_N; i += blockDim.x * blockDim.y) {
        int row = i / BLOCK_N;
        int col = i % BLOCK_N;
        if (row < K && bx * BLOCK_N + col < N) {
            Bs[write_buf][row][col] = ldg_double(&B_global[row * ldb + col]);
        } else {
            Bs[write_buf][row][col] = 0.0;
        }
    }
    
    __syncthreads();
    
    // Main K-loop with double buffering
    for (int k_tile = 0; k_tile < K; k_tile += BLOCK_K) {
        // Swap buffers
        read_buf = write_buf;
        write_buf = 1 - write_buf;
        
        // Prefetch next tile (if not last iteration)
        if (k_tile + BLOCK_K < K) {
            // Async load A tile for next iteration
            #pragma unroll 4
            for (int i = tid; i < BLOCK_M * BLOCK_K; i += blockDim.x * blockDim.y) {
                int row = i / BLOCK_K;
                int col = i % BLOCK_K;
                int global_k = k_tile + BLOCK_K + col;
                if (by * BLOCK_M + row < M && global_k < K) {
                    As[write_buf][row][col] = ldg_double(&A_global[row * lda + global_k]);
                } else {
                    As[write_buf][row][col] = 0.0;
                }
            }
            
            // Async load B tile for next iteration
            #pragma unroll 4
            for (int i = tid; i < BLOCK_K * BLOCK_N; i += blockDim.x * blockDim.y) {
                int row = i / BLOCK_N;
                int col = i % BLOCK_N;
                int global_k = k_tile + BLOCK_K + row;
                if (global_k < K && bx * BLOCK_N + col < N) {
                    Bs[write_buf][row][col] = ldg_double(&B_global[global_k * ldb + col]);
                } else {
                    Bs[write_buf][row][col] = 0.0;
                }
            }
        }
        
        // Compute on current tile with aggressive unrolling
        #pragma unroll
        for (int k = 0; k < BLOCK_K; k++) {
            // Prefetch from shared memory to registers
            double a_reg[THREAD_M];
            double b_reg[THREAD_N];
            
            // Load A values
            #pragma unroll
            for (int i = 0; i < THREAD_M; i++) {
                int row_idx = warpRow * 32 + threadRowInWarp + i;
                if (row_idx < BLOCK_M) {
                    a_reg[i] = As[read_buf][row_idx][k];
                } else {
                    a_reg[i] = 0.0;
                }
            }
            
            // Load B values
            #pragma unroll
            for (int j = 0; j < THREAD_N; j++) {
                int col_idx = warpCol * 64 + threadColInWarp + j;
                if (col_idx < BLOCK_N) {
                    b_reg[j] = Bs[read_buf][k][col_idx];
                } else {
                    b_reg[j] = 0.0;
                }
            }
            
            // Compute outer product with FMA
            #pragma unroll
            for (int i = 0; i < THREAD_M; i++) {
                #pragma unroll
                for (int j = 0; j < THREAD_N; j++) {
                    acc[i][j] = fma_double(a_reg[i], b_reg[j], acc[i][j]);
                }
            }
        }
        
        __syncthreads();
        
        // Move pointers for next iteration
        A_global += BLOCK_K;
        B_global += BLOCK_K * ldb;
    }
    
    // Write results back to global memory with coalesced access
    #pragma unroll
    for (int i = 0; i < THREAD_M; i++) {
        #pragma unroll
        for (int j = 0; j < THREAD_N; j++) {
            int row = globalRow + i;
            int col = globalCol + j;
            if (row < M && col < N) {
                int idx = row * ldc + col;
                if (beta == 0.0) {
                    C[idx] = alpha * acc[i][j];
                } else {
                    C[idx] = fma_double(alpha, acc[i][j], beta * C[idx]);
                }
            }
        }
    }
}

// Host-side GEMM reference
void gemm_host(int M, int N, int K, 
               double alpha, const double* A, int lda,
               const double* B, int ldb,
               double beta, double* C, int ldc) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            double sum = 0.0;
            for (int k = 0; k < K; k++) {
                sum += A[i * lda + k] * B[k * ldb + j];
            }
            C[i * ldc + j] = alpha * sum + beta * C[i * ldc + j];
        }
    }
}

void init_matrix_random(double* mat, int rows, int cols, int ld) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            mat[i * ld + j] = (double)rand() / RAND_MAX;
        }
    }
}

void init_matrix_constant(double* mat, int rows, int cols, int ld, double val) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            mat[i * ld + j] = val;
        }
    }
}

void copy_matrix(const double* src, double* dst, int rows, int cols, int ld) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            dst[i * ld + j] = src[i * ld + j];
        }
    }
}

double verify_result(const double* C_host, const double* C_device, int M, int N, int ldc) {
    double max_error = 0.0;
    double max_rel_error = 0.0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            int idx = i * ldc + j;
            double error = fabs(C_host[idx] - C_device[idx]);
            double rel_error = error / (fabs(C_host[idx]) + 1e-10);
            
            if (error > max_error) max_error = error;
            if (rel_error > max_rel_error) max_rel_error = rel_error;
        }
    }
    
    return max_rel_error;
}

double get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

int main() {
    printf("# GEMM CUDA Implementation v1.5.0\n");
    
    // Matrix dimensions
    const int M = 2048;
    const int N = 2048;
    const int K = 2048;
    const double alpha = 1.0;
    const double beta = 0.0;
    
    printf("# Matrix dimensions: M=%d, N=%d, K=%d\n", M, N, K);
    printf("# Data type: double precision (64-bit)\n");
    printf("# Optimization: Maximum register usage, aggressive unrolling, double buffering\n");
    printf("# Block configuration: M=%d, N=%d, K=%d\n", BLOCK_M, BLOCK_N, BLOCK_K);
    printf("# Thread tile: M=%d, N=%d\n", THREAD_M, THREAD_N);
    
    // Device information
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    printf("# Number of CUDA devices: %d\n", deviceCount);
    
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    printf("# Using GPU: %s\n", prop.name);
    printf("# Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("# Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("# SMs: %d\n", prop.multiProcessorCount);
    printf("# Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("# Shared memory per block: %zu bytes\n", prop.sharedMemPerBlock);
    printf("# Registers per block: %d\n", prop.regsPerBlock);
    
    // Set cache configuration
    CUDA_CHECK(hipDeviceSetCacheConfig(hipFuncCachePreferShared));
    
    // Host memory allocation
    double *h_A = (double*)malloc(M * K * sizeof(double));
    double *h_B = (double*)malloc(K * N * sizeof(double));
    double *h_C = (double*)malloc(M * N * sizeof(double));
    double *h_C_ref = (double*)malloc(M * N * sizeof(double));
    
    // Initialize matrices
    srand(42);
    init_matrix_random(h_A, M, K, K);
    init_matrix_random(h_B, K, N, N);
    init_matrix_constant(h_C, M, N, N, 0.0);
    copy_matrix(h_C, h_C_ref, M, N, N);
    
    // Device memory allocation
    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, M * N * sizeof(double)));
    
    // Transfer to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, M * K * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K * N * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, h_C, M * N * sizeof(double), hipMemcpyHostToDevice));
    
    // Kernel configuration
    dim3 blockDim(256);  // 256 threads per block
    dim3 gridDim((N + BLOCK_N - 1) / BLOCK_N, (M + BLOCK_M - 1) / BLOCK_M);
    
    printf("# Grid dimensions: %d x %d\n", gridDim.x, gridDim.y);
    printf("# Block dimensions: %d threads\n", blockDim.x);
    
    // Shared memory size calculation
    size_t shmem_size = 2 * (BLOCK_M * (BLOCK_K + 1) + BLOCK_K * (BLOCK_N + 1)) * sizeof(double);
    printf("# Shared memory usage: %zu bytes (double buffered with padding)\n", shmem_size);
    
    // Check shared memory limit
    if (shmem_size > prop.sharedMemPerBlock) {
        printf("# ERROR: Required shared memory (%zu) exceeds device limit (%zu)\n", 
               shmem_size, prop.sharedMemPerBlock);
        shmem_size = 0;  // Use dynamic allocation
    }
    
    // Warmup runs
    printf("\n# Warming up...\n");
    for (int i = 0; i < 3; i++) {
        CUDA_CHECK(hipMemcpy(d_C, h_C, M * N * sizeof(double), hipMemcpyHostToDevice));
        gemm_kernel_optimized<<<gridDim, blockDim, shmem_size>>>(M, N, K, alpha, d_A, K, d_B, N, beta, d_C, N);
        CUDA_CHECK(hipDeviceSynchronize());
    }
    
    // Performance measurement
    int n_trials = 5;
    double total_time = 0.0;
    double best_time = 1e10;
    
    printf("\n# Performance measurements:\n");
    
    for (int trial = 0; trial < n_trials; trial++) {
        CUDA_CHECK(hipMemcpy(d_C, h_C, M * N * sizeof(double), hipMemcpyHostToDevice));
        
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        
        CUDA_CHECK(hipEventRecord(start));
        gemm_kernel_optimized<<<gridDim, blockDim, shmem_size>>>(M, N, K, alpha, d_A, K, d_B, N, beta, d_C, N);
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        
        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        double seconds = milliseconds / 1000.0;
        
        double gflops = (2.0 * M * N * K) / (seconds * 1e9);
        printf("# Trial %d: %.3f ms, %.3f GFLOPS\n", trial + 1, milliseconds, gflops);
        
        if (seconds < best_time) {
            best_time = seconds;
        }
        
        if (trial > 0) {  // Skip first trial (additional warmup)
            total_time += seconds;
        }
        
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }
    
    // Get results
    CUDA_CHECK(hipMemcpy(h_C, d_C, M * N * sizeof(double), hipMemcpyDeviceToHost));
    
    // Accuracy verification
    const int verify_size = 64;
    double *h_A_small = (double*)malloc(verify_size * verify_size * sizeof(double));
    double *h_B_small = (double*)malloc(verify_size * verify_size * sizeof(double));
    double *h_C_small = (double*)malloc(verify_size * verify_size * sizeof(double));
    double *h_C_small_ref = (double*)malloc(verify_size * verify_size * sizeof(double));
    
    for (int i = 0; i < verify_size; i++) {
        for (int j = 0; j < verify_size; j++) {
            h_A_small[i * verify_size + j] = h_A[i * K + j];
            h_B_small[i * verify_size + j] = h_B[i * N + j];
            h_C_small[i * verify_size + j] = 0.0;
            h_C_small_ref[i * verify_size + j] = 0.0;
        }
    }
    
    // Host reference
    gemm_host(verify_size, verify_size, verify_size, alpha, h_A_small, verify_size, 
              h_B_small, verify_size, beta, h_C_small_ref, verify_size);
    
    // Device computation
    double *d_A_small, *d_B_small, *d_C_small;
    CUDA_CHECK(hipMalloc(&d_A_small, verify_size * verify_size * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B_small, verify_size * verify_size * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C_small, verify_size * verify_size * sizeof(double)));
    
    CUDA_CHECK(hipMemcpy(d_A_small, h_A_small, verify_size * verify_size * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_small, h_B_small, verify_size * verify_size * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_small, h_C_small, verify_size * verify_size * sizeof(double), hipMemcpyHostToDevice));
    
    dim3 verify_grid(1, 1);
    dim3 verify_block(256);
    gemm_kernel_optimized<<<verify_grid, verify_block>>>(verify_size, verify_size, verify_size, 
                                                         alpha, d_A_small, verify_size, 
                                                         d_B_small, verify_size, 
                                                         beta, d_C_small, verify_size);
    CUDA_CHECK(hipDeviceSynchronize());
    
    CUDA_CHECK(hipMemcpy(h_C_small, d_C_small, verify_size * verify_size * sizeof(double), hipMemcpyDeviceToHost));
    
    double max_rel_error = verify_result(h_C_small_ref, h_C_small, verify_size, verify_size, verify_size);
    
    // Final results
    double avg_time = total_time / (n_trials - 1);
    double avg_gflops = (2.0 * M * N * K) / (avg_time * 1e9);
    double best_gflops = (2.0 * M * N * K) / (best_time * 1e9);
    
    printf("\n# Final Results:\n");
    printf("# Best time: %.3f ms\n", best_time * 1000);
    printf("# Best performance: %.3f GFLOPS\n", best_gflops);
    printf("# Average time: %.3f ms\n", avg_time * 1000);
    printf("# Average performance: %.3f GFLOPS\n", avg_gflops);
    printf("# Max relative error: %.2e\n", max_rel_error);
    
    // V100 theoretical peak (7.8 TFLOPS for FP64)
    double efficiency = (best_gflops / 7800.0) * 100.0;
    printf("# Efficiency vs theoretical peak: %.2f%%\n", efficiency);
    
    // Accuracy check
    if (max_rel_error < 1e-9) {
        printf("# Accuracy: PASS (relative error < 1e-9)\n");
    } else {
        printf("# Accuracy: FAIL (relative error = %.2e)\n", max_rel_error);
    }
    
    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);
    free(h_A_small);
    free(h_B_small);
    free(h_C_small);
    free(h_C_small_ref);
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_A_small));
    CUDA_CHECK(hipFree(d_B_small));
    CUDA_CHECK(hipFree(d_C_small));
    
    return 0;
}