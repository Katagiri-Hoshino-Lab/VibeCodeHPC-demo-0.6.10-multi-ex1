// GEMM CUDA v1.5.1 - Fixed boundary conditions and indexing
// Corrected thread mapping and memory access patterns
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", \
                __FILE__, __LINE__, hipGetErrorString(error)); \
        exit(1); \
    } \
} while(0)

// v1.5.1: Corrected tile sizes and thread mapping
#define BLOCK_M 64
#define BLOCK_N 64  
#define BLOCK_K 16
#define THREAD_M 4
#define THREAD_N 4

// Read-only cache optimization
__device__ __forceinline__ double ldg_double(const double* ptr) {
#if __CUDA_ARCH__ >= 350
    return __ldg(ptr);
#else
    return *ptr;
#endif
}

// FMA optimization
__device__ __forceinline__ double fma_double(double a, double b, double c) {
    return fma(a, b, c);
}

__global__ void gemm_kernel_optimized(
    int M, int N, int K,
    double alpha, const double* __restrict__ A, int lda,
    const double* __restrict__ B, int ldb,
    double beta, double* __restrict__ C, int ldc) {
    
    // Double buffering for overlapping computation and memory access
    __shared__ double As[2][BLOCK_M][BLOCK_K];
    __shared__ double Bs[2][BLOCK_K][BLOCK_N];
    
    // Simple thread indexing
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    
    // Block position
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    
    // Thread tile configuration (16x16 threads, each handles 4x4)
    const int threads_per_dim = 16;
    const int thread_row = ty;
    const int thread_col = tx;
    
    // Starting position for this thread's tile
    const int c_row_start = by * BLOCK_M + thread_row * THREAD_M;
    const int c_col_start = bx * BLOCK_N + thread_col * THREAD_N;
    
    // Register file for accumulation
    double acc[THREAD_M][THREAD_N];
    
    // Initialize accumulators
    #pragma unroll
    for (int i = 0; i < THREAD_M; i++) {
        #pragma unroll
        for (int j = 0; j < THREAD_N; j++) {
            acc[i][j] = 0.0;
        }
    }
    
    // Double buffering indices
    int write_stage = 0;
    int read_stage = 1;
    
    // Load first tile to shared memory
    // Each thread loads multiple elements for coalescing
    #pragma unroll
    for (int i = tid; i < BLOCK_M * BLOCK_K; i += blockDim.x * blockDim.y) {
        int row = i / BLOCK_K;
        int col = i % BLOCK_K;
        int global_row = by * BLOCK_M + row;
        
        if (global_row < M && col < K) {
            As[write_stage][row][col] = A[global_row * lda + col];
        } else {
            As[write_stage][row][col] = 0.0;
        }
    }
    
    #pragma unroll
    for (int i = tid; i < BLOCK_K * BLOCK_N; i += blockDim.x * blockDim.y) {
        int row = i / BLOCK_N;
        int col = i % BLOCK_N;
        int global_col = bx * BLOCK_N + col;
        
        if (row < K && global_col < N) {
            Bs[write_stage][row][col] = B[row * ldb + global_col];
        } else {
            Bs[write_stage][row][col] = 0.0;
        }
    }
    
    __syncthreads();
    
    // Main K-loop with double buffering
    for (int k_tile = 0; k_tile < K; k_tile += BLOCK_K) {
        // Swap buffers
        read_stage = write_stage;
        write_stage = 1 - write_stage;
        
        // Prefetch next tile (if not last iteration)
        if (k_tile + BLOCK_K < K) {
            // Load A tile for next iteration
            #pragma unroll
            for (int i = tid; i < BLOCK_M * BLOCK_K; i += blockDim.x * blockDim.y) {
                int row = i / BLOCK_K;
                int col = i % BLOCK_K;
                int global_row = by * BLOCK_M + row;
                int global_k = k_tile + BLOCK_K + col;
                
                if (global_row < M && global_k < K) {
                    As[write_stage][row][col] = ldg_double(&A[global_row * lda + global_k]);
                } else {
                    As[write_stage][row][col] = 0.0;
                }
            }
            
            // Load B tile for next iteration
            #pragma unroll
            for (int i = tid; i < BLOCK_K * BLOCK_N; i += blockDim.x * blockDim.y) {
                int row = i / BLOCK_N;
                int col = i % BLOCK_N;
                int global_col = bx * BLOCK_N + col;
                int global_k = k_tile + BLOCK_K + row;
                
                if (global_k < K && global_col < N) {
                    Bs[write_stage][row][col] = ldg_double(&B[global_k * ldb + global_col]);
                } else {
                    Bs[write_stage][row][col] = 0.0;
                }
            }
        }
        
        // Compute on current tile
        #pragma unroll
        for (int k = 0; k < BLOCK_K; k++) {
            // Load values to registers
            double a_reg[THREAD_M];
            double b_reg[THREAD_N];
            
            #pragma unroll
            for (int i = 0; i < THREAD_M; i++) {
                int row_idx = thread_row * THREAD_M + i;
                a_reg[i] = As[read_stage][row_idx][k];
            }
            
            #pragma unroll
            for (int j = 0; j < THREAD_N; j++) {
                int col_idx = thread_col * THREAD_N + j;
                b_reg[j] = Bs[read_stage][k][col_idx];
            }
            
            // Compute outer product with FMA
            #pragma unroll
            for (int i = 0; i < THREAD_M; i++) {
                #pragma unroll
                for (int j = 0; j < THREAD_N; j++) {
                    acc[i][j] = fma_double(a_reg[i], b_reg[j], acc[i][j]);
                }
            }
        }
        
        __syncthreads();
    }
    
    // Write results back to global memory
    #pragma unroll
    for (int i = 0; i < THREAD_M; i++) {
        #pragma unroll
        for (int j = 0; j < THREAD_N; j++) {
            int global_row = c_row_start + i;
            int global_col = c_col_start + j;
            
            if (global_row < M && global_col < N) {
                if (beta == 0.0) {
                    C[global_row * ldc + global_col] = alpha * acc[i][j];
                } else {
                    C[global_row * ldc + global_col] = 
                        alpha * acc[i][j] + beta * C[global_row * ldc + global_col];
                }
            }
        }
    }
}

// Host-side GEMM function for verification
void gemm_host(int M, int N, int K, 
               double alpha, const double* A, int lda,
               const double* B, int ldb,
               double beta, double* C, int ldc) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            double sum = 0.0;
            for (int k = 0; k < K; k++) {
                sum += A[i * lda + k] * B[k * ldb + j];
            }
            C[i * ldc + j] = alpha * sum + beta * C[i * ldc + j];
        }
    }
}

// Matrix initialization functions
void init_matrix_random(double* mat, int rows, int cols, int ld) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            mat[i * ld + j] = (double)rand() / RAND_MAX;
        }
    }
}

void init_matrix_constant(double* mat, int rows, int cols, int ld, double val) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            mat[i * ld + j] = val;
        }
    }
}

void copy_matrix(const double* src, double* dst, int rows, int cols, int ld) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            dst[i * ld + j] = src[i * ld + j];
        }
    }
}

// Result verification
double verify_result(const double* C_host, const double* C_device, int M, int N, int ldc) {
    double max_error = 0.0;
    double max_rel_error = 0.0;
    
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            int idx = i * ldc + j;
            double error = fabs(C_host[idx] - C_device[idx]);
            double rel_error = error / (fabs(C_host[idx]) + 1e-10);
            
            if (error > max_error) max_error = error;
            if (rel_error > max_rel_error) max_rel_error = rel_error;
        }
    }
    
    return max_rel_error;
}

double get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

int main() {
    printf("# GEMM CUDA Implementation v1.5.1\n");
    
    // Matrix dimensions
    const int M = 2048;
    const int N = 2048;
    const int K = 2048;
    const double alpha = 1.0;
    const double beta = 0.0;
    
    printf("# Matrix dimensions: M=%d, N=%d, K=%d\n", M, N, K);
    printf("# Data type: double precision (64-bit)\n");
    printf("# Optimization: Fixed indexing and boundary checks\n");
    printf("# Block size: %dx%d, Block K: %d\n", BLOCK_M, BLOCK_N, BLOCK_K);
    printf("# Thread tile: %dx%d\n", THREAD_M, THREAD_N);
    
    // Device information
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));
    printf("# Number of CUDA devices: %d\n", deviceCount);
    
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    printf("# Using GPU: %s\n", prop.name);
    printf("# Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("# Memory: %.2f GB\n", prop.totalGlobalMem / (1024.0 * 1024 * 1024));
    printf("# SMs: %d\n", prop.multiProcessorCount);
    
    // Host memory allocation
    double *h_A = (double*)malloc(M * K * sizeof(double));
    double *h_B = (double*)malloc(K * N * sizeof(double));
    double *h_C = (double*)malloc(M * N * sizeof(double));
    double *h_C_ref = (double*)malloc(M * N * sizeof(double));
    
    // Matrix initialization
    srand(42);
    init_matrix_random(h_A, M, K, K);
    init_matrix_random(h_B, K, N, N);
    init_matrix_constant(h_C, M, N, N, 0.0);
    copy_matrix(h_C, h_C_ref, M, N, N);
    
    // Device memory allocation
    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A, M * K * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B, K * N * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C, M * N * sizeof(double)));
    
    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_A, h_A, M * K * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, K * N * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C, h_C, M * N * sizeof(double), hipMemcpyHostToDevice));
    
    // Kernel configuration - 16x16 threads per block
    dim3 blockDim(16, 16);
    dim3 gridDim((N + BLOCK_N - 1) / BLOCK_N, (M + BLOCK_M - 1) / BLOCK_M);
    
    printf("# Grid: %dx%d, Block: %dx%d\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);
    
    // Warmup
    printf("\n# Warming up...\n");
    for (int i = 0; i < 3; i++) {
        CUDA_CHECK(hipMemcpy(d_C, h_C, M * N * sizeof(double), hipMemcpyHostToDevice));
        gemm_kernel_optimized<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, K, d_B, N, beta, d_C, N);
        CUDA_CHECK(hipDeviceSynchronize());
    }
    
    // Performance measurement
    int n_trials = 10;
    double total_time = 0.0;
    double best_time = 1e10;
    double best_gflops = 0.0;
    
    printf("\n# Performance measurements:\n");
    
    for (int trial = 0; trial < n_trials; trial++) {
        CUDA_CHECK(hipMemcpy(d_C, h_C, M * N * sizeof(double), hipMemcpyHostToDevice));
        
        hipEvent_t start, stop;
        CUDA_CHECK(hipEventCreate(&start));
        CUDA_CHECK(hipEventCreate(&stop));
        
        CUDA_CHECK(hipEventRecord(start));
        
        gemm_kernel_optimized<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, K, d_B, N, beta, d_C, N);
        
        CUDA_CHECK(hipEventRecord(stop));
        CUDA_CHECK(hipEventSynchronize(stop));
        
        float milliseconds = 0;
        CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
        double seconds = milliseconds / 1000.0;
        
        double gflops = (2.0 * M * N * K) / (seconds * 1e9);
        printf("# Trial %2d: %.3f ms, %.3f GFLOPS\n", trial + 1, milliseconds, gflops);
        
        if (seconds < best_time) {
            best_time = seconds;
            best_gflops = gflops;
        }
        
        if (trial >= 2) {  // Skip first 2 trials (warmup)
            total_time += seconds;
        }
        
        CUDA_CHECK(hipEventDestroy(start));
        CUDA_CHECK(hipEventDestroy(stop));
    }
    
    // Get results
    CUDA_CHECK(hipMemcpy(h_C, d_C, M * N * sizeof(double), hipMemcpyDeviceToHost));
    
    // Accuracy verification (small-scale)
    const int verify_size = 64;
    double *h_A_small = (double*)malloc(verify_size * verify_size * sizeof(double));
    double *h_B_small = (double*)malloc(verify_size * verify_size * sizeof(double));
    double *h_C_small = (double*)malloc(verify_size * verify_size * sizeof(double));
    double *h_C_small_ref = (double*)malloc(verify_size * verify_size * sizeof(double));
    
    // Initialize small matrices for verification
    for (int i = 0; i < verify_size; i++) {
        for (int j = 0; j < verify_size; j++) {
            h_A_small[i * verify_size + j] = h_A[i * K + j];
            h_B_small[i * verify_size + j] = h_B[i * N + j];
            h_C_small[i * verify_size + j] = 0.0;
            h_C_small_ref[i * verify_size + j] = 0.0;
        }
    }
    
    // Host reference implementation
    gemm_host(verify_size, verify_size, verify_size, alpha, h_A_small, verify_size, 
              h_B_small, verify_size, beta, h_C_small_ref, verify_size);
    
    // Device computation
    double *d_A_small, *d_B_small, *d_C_small;
    CUDA_CHECK(hipMalloc(&d_A_small, verify_size * verify_size * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_B_small, verify_size * verify_size * sizeof(double)));
    CUDA_CHECK(hipMalloc(&d_C_small, verify_size * verify_size * sizeof(double)));
    
    CUDA_CHECK(hipMemcpy(d_A_small, h_A_small, verify_size * verify_size * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B_small, h_B_small, verify_size * verify_size * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_small, h_C_small, verify_size * verify_size * sizeof(double), hipMemcpyHostToDevice));
    
    dim3 verify_grid(1, 1);
    dim3 verify_block(16, 16);
    gemm_kernel_optimized<<<verify_grid, verify_block>>>(
        verify_size, verify_size, verify_size, alpha, d_A_small, verify_size, 
        d_B_small, verify_size, beta, d_C_small, verify_size);
    CUDA_CHECK(hipDeviceSynchronize());
    
    CUDA_CHECK(hipMemcpy(h_C_small, d_C_small, verify_size * verify_size * sizeof(double), hipMemcpyDeviceToHost));
    
    double max_rel_error = verify_result(h_C_small_ref, h_C_small, verify_size, verify_size, verify_size);
    
    // Final results
    double avg_time = total_time / (n_trials - 2);
    double avg_gflops = (2.0 * M * N * K) / (avg_time * 1e9);
    
    printf("\n# Final Results:\n");
    printf("# Best time: %.3f ms\n", best_time * 1000);
    printf("# Best performance: %.3f GFLOPS\n", best_gflops);
    printf("# Average time (excluding warmup): %.3f ms\n", avg_time * 1000);
    printf("# Average performance: %.3f GFLOPS\n", avg_gflops);
    printf("# Max relative error: %.2e\n", max_rel_error);
    
    // V100 theoretical performance (7.8 TFLOPS for FP64)
    double efficiency = (best_gflops / 7800.0) * 100.0;
    printf("# Efficiency vs theoretical peak: %.2f%%\n", efficiency);
    
    // Accuracy check
    if (max_rel_error < 1e-9) {
        printf("# Accuracy: PASS (relative error < 1e-9)\n");
    } else {
        printf("# Accuracy: FAIL (relative error = %.2e)\n", max_rel_error);
    }
    
    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);
    free(h_A_small);
    free(h_B_small);
    free(h_C_small);
    free(h_C_small_ref);
    
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_A_small));
    CUDA_CHECK(hipFree(d_B_small));
    CUDA_CHECK(hipFree(d_C_small));
    
    return 0;
}